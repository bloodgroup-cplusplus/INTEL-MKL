#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>
#include<stdlib.h>
#include<math.h>
#include<assert.h>
#include<iostream>
// cuda kernal for vector addition
__global__  void vectorAdd(int* a, int* b, int* c, int n)
{
	// Calculate global thread ID(tid)
	// one thread per element that gets added.
	//out of all of our thread figure out who am I 
	// blockIdx.x => which block am i 
	//bolckDim.x => block size ( 256)
	// threadIdx.x => which thread am I in the block.
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	// vector boundary guard
	if (tid < n)
	{
		//Each thread adds a single element
		c[tid] = a[tid] + b[tid];
	}
}

// Initialize vector of size n to int between 0-99

void matrix_init(int* a, int n)
{
	for (int i = 0; i < n; ++i)
	{
		a[i] = rand() % 100;
	}
}
//Check vector add result
void error_check(int* a, int* b, int* c, int n)
{
	for (int i = 0; i < n; ++i)
	{
		assert(c[i] == a[i] + b[i]);
	}
}

int main()
{
	// vector size of 2^16 (65536 elements)
	int n = 1 << 16;

	// Host vector pointers

	int* h_a, * h_b, * h_c;
	//Device vector pointers

	int* d_a, * d_b, * d_c;
	//Allocation size for all vectors

	size_t bytes = sizeof(int) * n;

	//Allocate host memory

	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	//Allocate device memory

	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);
	//Initialize vectors a and b with random values between  0 and 99;
	matrix_init(h_a, n);
	matrix_init(h_b, n);

	// Threadblock size

	int NUM_THREADS = 256;

	//Grid size
	int NUM_BLOCKS = (int)ceil(n / NUM_THREADS);

	//Launch kernel on default strream w/o stream

	vectorAdd << < NUM_BLOCKS, NUM_THREADS >> > (d_a, d_b, d_c, n);


	// copy sum vector from device to host

	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);


	//check result for errors


	error_check(h_a, h_b, h_c, n);


	std::cout << "COMPLETED SUCCESSUFLLY\n" << std::endl;
	return 0;


}
